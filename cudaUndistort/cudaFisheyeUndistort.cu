#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

using namespace std;

__global__ void cudaBuildMap(float *pCamK, float *pDistort, float *pInvNewCamK, float *pMapx, float *pMapy, int outImgW, int outImgH)
{
	const int tidx = blockDim.x*blockIdx.x + threadIdx.x;
	const int tidy = blockDim.y*blockIdx.y + threadIdx.y;
	if (tidx < outImgW && tidy < outImgH)
	{
        //ͼ������ϵ->���������ϵ
		float _x = tidx*pInvNewCamK[0] + tidy*pInvNewCamK[1] + pInvNewCamK[2];
		float _y = tidx*pInvNewCamK[3] + tidy*pInvNewCamK[4] + pInvNewCamK[5];
		float _w = tidx*pInvNewCamK[6] + tidy*pInvNewCamK[7] + pInvNewCamK[8];
        //��һ��
		float x = _x / _w;
		float y = _y / _w;
        //����У��
		float r = sqrt(x*x + y*y);
		float theta = atan(r);

		float theta2 = theta*theta;
		float theta4 = theta2*theta2;
		float theta6 = theta4*theta2;
		float theta8 = theta4*theta4;
		float theta_d = theta * (1 + pDistort[0] * theta2 + pDistort[1] * theta4 + pDistort[2] * theta6 + pDistort[3] * theta8);
        //��ͶӰ��ͼ������ϵ
		float scale = (r == 0) ? 1.0 : theta_d / r;
		float u = pCamK[0] * x * scale + pCamK[2];
		float v = pCamK[4] * y * scale + pCamK[5];
        //����Map
		int mapIdx = tidy*outImgW + tidx;
		pMapx[mapIdx] = (float)u;
		pMapy[mapIdx] = (float)v;
	}
}

__global__ void cudaRemap (uchar* pSrcImg, uchar* pDstImg, float* pMapx, float* pMapy, int inWidth, int inHeight, int outWidth, int outHeight, int channels)
{
	const int tidx = blockDim.x*blockIdx.x + threadIdx.x;
	const int tidy = blockDim.y*blockIdx.y + threadIdx.y;
	if (tidx < outWidth && tidy < outHeight)
	{
		int mapIdx = tidy*outWidth + tidx;
		float u = pMapx[mapIdx];
		float v = pMapy[mapIdx];
        //˫���Բ�ֵ
		int u1 = floor(u);
		int v1 = floor(v);
		int u2 = u1 + 1;
		int v2 = v1 + 1;
		if (u1 >= 0 && v1 >= 0 && u2 < inWidth && v2 < inHeight)
		{
			float dx = u - u1;
			float dy = v - v1;
			float weight1 = (1 - dx)*(1 - dy);
			float weight2 = dx*(1 - dy);
			float weight3 = (1 - dx)*dy;
			float weight4 = dx*dy;

			int resultIdx = mapIdx * 3;
			for (int chan = 0; chan < channels; chan++)
			{
				pDstImg[resultIdx + chan] = uchar(weight1*pSrcImg[(v1*inWidth + u1) * 3 + chan]
					+ weight2*pSrcImg[(v1*inWidth + u2) * 3 + chan]
					+ weight3*pSrcImg[(v2*inWidth + u1) * 3 + chan]
					+ weight4*pSrcImg[(v2*inWidth + u2) * 3 + chan]);
			}
		}
	}
}

void main()
{
	cv::Mat srcImg = cv::imread("data/srcImg/1.jpg");
	cv::Mat camK = cv::Mat::eye(3, 3, CV_32F);
	cv::Mat newCamK = cv::Mat::eye(3, 3, CV_32F);
	cv::Mat invCamK = cv::Mat::eye(3, 3, CV_32F);
	cv::Mat D = cv::Mat::zeros(4, 1, CV_32F);
	cv::Mat R = cv::Mat::eye(3, 3, CV_32F);
	int imgHeight = srcImg.rows;
	int imgWidth = srcImg.cols;
	int channels = srcImg.channels();
	int outImgHeight = imgHeight;
	int outImgWidth = imgWidth;
	cv::Mat undistortImg=cv::Mat(outImgHeight, outImgWidth, CV_8UC3);
	cv::Mat mapx = cv::Mat(outImgHeight, outImgWidth, CV_32F);
	cv::Mat mapy = cv::Mat(outImgHeight, outImgWidth, CV_32F);

	//�ڲξ���
	camK.at<float>(0, 0) = 1212.76;
	camK.at<float>(1, 1) = 1211.92;
	camK.at<float>(0, 2) = 2017.81;
	camK.at<float>(1, 2) = 1483.59;
	//����ϵ������
	float disCoeff[4] = { -0.0217375,-0.00271412, 2.81773e-005,-0.000196795 };
	D = cv::Mat(4, 1, CV_32F, disCoeff);
	//�ڲξ�������
	newCamK = camK.clone();
	invCamK = newCamK.inv(cv::DECOMP_SVD);

    //����GPU�ڴ沢�ϴ�������GPU
	hipError_t err;
	float *pCamKData = NULL;
	float *pInvNewCamKData = NULL;
	float *pDistortData = NULL;
	uchar *pSrcImgData = NULL;
	uchar *pDstImgData = NULL;
	float *pMapxData = NULL;
	float *pMapyData = NULL;
	{
		err = hipMalloc(&pCamKData, 9 * sizeof(float));
		err = hipMalloc(&pInvNewCamKData, 9 * sizeof(float));
		err = hipMalloc(&pDistortData, 4 * sizeof(float));
		err = hipMalloc(&pSrcImgData, imgHeight*imgWidth * sizeof(uchar)*channels);
		err = hipMalloc(&pMapxData, outImgHeight*outImgWidth * sizeof(float));
		err = hipMalloc(&pMapyData, outImgHeight*outImgWidth * sizeof(float));
		err = hipMalloc(&pDstImgData, outImgHeight*outImgWidth * sizeof(uchar)*channels);
	}
	{
		err = hipMemcpy(pCamKData, camK.data, 9 * sizeof(float), hipMemcpyHostToDevice);
		err = hipMemcpy(pInvNewCamKData, invCamK.data, 9 * sizeof(float), hipMemcpyHostToDevice);
		err = hipMemcpy(pDistortData, D.data, 4 * sizeof(float), hipMemcpyHostToDevice);
		err = hipMemcpy(pMapxData, mapx.data, outImgHeight*outImgWidth * sizeof(float), hipMemcpyHostToDevice);
		err = hipMemcpy(pMapyData, mapy.data, outImgHeight*outImgWidth * sizeof(float), hipMemcpyHostToDevice);
		err = hipMemcpy(pSrcImgData, srcImg.data, imgHeight*imgWidth * sizeof(uchar)*channels, hipMemcpyHostToDevice);
	}

	dim3 block(16, 16);
	dim3 grid((imgWidth + block.x - 1) / block.x, (imgHeight + block.y - 1) / block.y);
    //����Map
	cudaBuildMap << <grid, block >> > (pCamKData, pDistortData, pInvNewCamKData, pMapxData, pMapyData, outImgWidth, outImgHeight);
	hipDeviceSynchronize();
    //Remap
	cudaRemap << <grid, block >> > (pSrcImgData, pDstImgData, pMapxData, pMapyData, imgWidth, imgHeight, outImgWidth, outImgHeight, channels);
	hipDeviceSynchronize();
	err = hipGetLastError();
    //��������
	err = hipMemcpy(undistortImg.data, pDstImgData, outImgHeight*outImgWidth * sizeof(uchar)*channels, hipMemcpyDeviceToHost);
	
	cv::imwrite("data/undistortImg.jpg", undistortImg);
}